#include <hip/hip_runtime.h>
// SiLU activation function: SiLU(x) = x / (1 + exp(-x))
// Review on leetGPU.
__global__ void silu_kernel(const float* input, float* output, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        output[i] = input[i]/(1.0f + __expf(-input[i]));
    }
}

// input, output are device pointers
extern "C" void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    silu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}


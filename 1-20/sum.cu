#include <hip/hip_runtime.h>

// kernel for summing all threads in block
__global__ void reduce(const float* input, float* blockSum, int N) {
    extern __shared__ float smem[];
    int tid = threadIdx.x;


    // First grid stride sum, get values into shared memory
    float sum = 0.0f;
    for (int i = blockDim.x * blockIdx.x * 2 + tid; i < N; i+= gridDim.x*blockDim.x*2) {
        sum += input[i];
        if (i + blockDim.x < N) {
            sum += input[i+blockDim.x];
        } 
    }
    smem[tid] = sum;
    __syncthreads();
    // For loop reducing the size of the stride each iteration until full warp remains (32)
    // Alt (int stride = blockdim.x >> 1; stride > 0 stride >>= 1)
    for (int stride = blockDim.x / 2; stride >= warpSize; stride /= 2) {
        if (tid < stride) {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }
    // Warp shuffle tail (avoid _synchtreads when not needed)
    float val = smem[tid];
    if (tid < warpSize) {
        unsigned mask = __activemask();
        val += __shfl_down_sync(mask, val, 16);
        val += __shfl_down_sync(mask, val, 8);
        val += __shfl_down_sync(mask, val, 4);
        val += __shfl_down_sync(mask, val, 2);
        val += __shfl_down_sync(mask, val, 1);
    }
    // write result to blockSum
    if (tid == 0) {
        blockSum[blockIdx.x] = val;
    }
}


// input, output are device pointers
extern "C" void solve(const float* input, float* output, int N) {  
    int threadsPerBlock = 256;
    int numBlocks = 4096;
    //(threadsPerBlock*2 + N - 1) / (threadsPerBlock * 2); 
    size_t sharedBytes = threadsPerBlock * sizeof(float);

    float* blockSum = nullptr;
    hipMalloc(&blockSum, numBlocks*sizeof(float));
    float* h_blockSum = (float*)malloc(numBlocks * sizeof(float));
    //float h_blockSum[numBlocks];
    // first reduce to one result per Block
    reduce<<<numBlocks, threadsPerBlock, sharedBytes>>>(input, blockSum, N);
    // sum up the resulting sum in each block on CPU
    hipMemcpy(h_blockSum, blockSum, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    double acc = 0.0;
    for (int i=0; i<numBlocks; i++) {
        acc += (double)h_blockSum[i];
    }
    float result = (float)acc;
    hipMemcpy(output, &result, sizeof(float), hipMemcpyHostToDevice);
    // cleanup
    hipFree(blockSum);
    free(h_blockSum);
}


